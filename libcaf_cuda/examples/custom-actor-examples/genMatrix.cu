

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
//generate_random_matrix
extern "C" __global__
void generate_random_matrix(int* matrix, int total_elements, int seed, int max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    hiprandState state;
    hiprand_init((unsigned long long)seed, idx, 0, &state);

    unsigned int r = hiprand(&state);
    matrix[idx] = r % max_val;
}


