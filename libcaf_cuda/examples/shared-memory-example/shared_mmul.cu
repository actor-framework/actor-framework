
#include <hip/hip_runtime.h>
extern "C" __global__
void matrixMul(const int* __restrict__ a,
               const int* __restrict__ b,
               int* __restrict__ c,
               int N)
{
    const int TILE = 32;
    int row = blockIdx.y * blockDim.y + threadIdx.y; // global row in C
    int col = blockIdx.x * blockDim.x + threadIdx.x; // global col in C

    __shared__ int s_a[TILE * TILE];
    __shared__ int s_b[TILE * TILE];

    int acc = 0;

    // Sweep tiles across the K dimension
    for (int i = 0; i < N; i += TILE) {
        int aCol = i + threadIdx.x;
        int bRow = i + threadIdx.y;

        if (row < N && aCol < N)
            s_a[threadIdx.y * TILE + threadIdx.x] = a[row * N + aCol];
        else
            s_a[threadIdx.y * TILE + threadIdx.x] = 0;

        if (bRow < N && col < N)
            s_b[threadIdx.y * TILE + threadIdx.x] = b[bRow * N + col];
        else
            s_b[threadIdx.y * TILE + threadIdx.x] = 0;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE; ++k) {
            acc += s_a[threadIdx.y * TILE + k] *
                   s_b[k * TILE + threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        c[row * N + col] = acc;
}

extern "C" __global__
void matrixMulFloat(const float* __restrict__ a,
                    const float* __restrict__ b,
                    float* __restrict__ c,
                    int N)
{
    const int TILE = 32;
    int row = blockIdx.y * blockDim.y + threadIdx.y; // global row in C
    int col = blockIdx.x * blockDim.x + threadIdx.x; // global col in C

    __shared__ float s_a[TILE * TILE];
    __shared__ float s_b[TILE * TILE];

    float acc = 0.0f;

    // Sweep tiles across the K dimension
    for (int i = 0; i < N; i += TILE) {
        int aCol = i + threadIdx.x;
        int bRow = i + threadIdx.y;

        if (row < N && aCol < N)
            s_a[threadIdx.y * TILE + threadIdx.x] = a[row * N + aCol];
        else
            s_a[threadIdx.y * TILE + threadIdx.x] = 0.0f;

        if (bRow < N && col < N)
            s_b[threadIdx.y * TILE + threadIdx.x] = b[bRow * N + col];
        else
            s_b[threadIdx.y * TILE + threadIdx.x] = 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE; ++k) {
            acc += s_a[threadIdx.y * TILE + k] *
                   s_b[k * TILE + threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        c[row * N + col] = acc;
}

