
#include <hip/hip_runtime.h>
extern "C" __global__
void matrixMul(const int* __restrict__ a,
               const int* __restrict__ b,
               int* __restrict__ c,
               int N)
{
    const int TILE = 32;
    int row = blockIdx.y * blockDim.y + threadIdx.y; // global row in C
    int col = blockIdx.x * blockDim.x + threadIdx.x; // global col in C

    __shared__ int s_a[TILE * TILE];
    __shared__ int s_b[TILE * TILE];

    int acc = 0;

    // Sweep tiles across the K dimension
    for (int i = 0; i < N; i += TILE) {

        // Each thread loads one element into shared memory (with bounds checks)
        int aCol = i + threadIdx.x;
        int bRow = i + threadIdx.y;

        // s_a[y, x] = a[row, aCol] if in range, else 0
        if (row < N && aCol < N)
            s_a[threadIdx.y * TILE + threadIdx.x] = a[row * N + aCol];
        else
            s_a[threadIdx.y * TILE + threadIdx.x] = 0;

        // s_b[y, x] = b[bRow, col] if in range, else 0
        if (bRow < N && col < N)
            s_b[threadIdx.y * TILE + threadIdx.x] = b[bRow * N + col];
        else
            s_b[threadIdx.y * TILE + threadIdx.x] = 0;

        __syncthreads();

        // Compute partial dot product for this tile
        #pragma unroll
        for (int k = 0; k < TILE; ++k) {
            acc += s_a[threadIdx.y * TILE + k] *
                   s_b[k * TILE + threadIdx.x];
        }

        __syncthreads();
    }

    // Final write (guarded)
    if (row < N && col < N)
        c[row * N + col] = acc;
}

