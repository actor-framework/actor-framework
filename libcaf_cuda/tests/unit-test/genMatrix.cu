

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
//generate_random_matrix
extern "C" __global__
void generate_random_matrix(int* matrix, int total_elements, int seed, int max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    hiprandState state;
    hiprand_init((unsigned long long)seed, idx, 0, &state);

    unsigned int r = hiprand(&state);
    matrix[idx] = r % max_val;
}


extern "C" __global__
void generate_random_matrix_float(float* matrix, int total_elements, int seed, float max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    hiprandState state;
    hiprand_init((unsigned long long)seed, idx, 0, &state);

    unsigned int r = hiprand(&state);
    matrix[idx] = (float)(r % (unsigned int)max_val);
}


extern "C" __global__
void generate_random_matrix_double(double* matrix, int total_elements, int seed, double max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    hiprandState state;
    hiprand_init((unsigned long long)seed, idx, 0, &state);

    unsigned int r = hiprand(&state);
    matrix[idx] = (double)(r % (unsigned int)max_val);
}



