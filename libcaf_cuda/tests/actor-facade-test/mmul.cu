
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
// mmul.cu
extern "C" __global__
void matrixMul(const int* a, const int* b, int* c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("%d\n",N);
    if (row < N && col < N) {
        int temp = 0;
        for (int k = 0; k < N; ++k) {
            temp += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = temp;
    }
}


//generate_random_matrix
extern "C" __global__
void generate_random_matrix(int* matrix, int total_elements, int seed, int max_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_elements) return;

    hiprandState state;
    hiprand_init((unsigned long long)seed, idx, 0, &state);

    unsigned int r = hiprand(&state);
    matrix[idx] = r % max_val;
}


